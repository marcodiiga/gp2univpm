#include "hip/hip_runtime.h"
//  Copyright 2015 Alesiani Marco <marco.diiga@gmail.com>
//
//  Licensed under the Apache License, Version 2.0 (the "License");
//  you may not use this file except in compliance with the License.
//  You may obtain a copy of the License at
//
//  http://www.apache.org/licenses/LICENSE-2.0
//
//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.
//

//  Simple GPU convolution with a gaussian blur kernel
//
#include <hip/hip_runtime.h>
#include <PPMReader.h>
#include <iostream>
#include <numeric>

// Can be calculated with custom parameters as follows:
// exp( -0.5 * (pow((x-mean)/sigma, 2.0) + pow((y-mean)/sigma,2.0)) ) / (2 * M_PI * sigma * sigma)
#define KERNEL_WIDTH 5
#define KERNEL_HEIGHT 5
std::vector<float> gaussianKernel = { // generated with sigma^2 == 3
     0.01905031014488527f, 0.03140865154930652f, 0.03710493756184187f, 0.03140865154930652f, 0.01905031014488527f,
     0.03140865154930652f, 0.05178411189334978f, 0.06117569980620832f, 0.05178411189334978f, 0.03140865154930652f,
     0.03710493756184187f, 0.06117569980620832f, 0.07227054998040688f, 0.06117569980620832f, 0.03710493756184187f,
     0.03140865154930652f, 0.05178411189334978f, 0.06117569980620832f, 0.05178411189334978f, 0.03140865154930652f,
     0.01905031014488527f, 0.03140865154930652f, 0.03710493756184187f, 0.03140865154930652f, 0.01905031014488527f 
};

// GPU constant memory
__constant__ float constantKernel[KERNEL_HEIGHT * KERNEL_WIDTH];
__constant__ float constantKernelSum;

__global__ void convolutionGPUKernel(unsigned char* image, unsigned char* resultImage, int rawImageWidth, int rawImageHeight, int pixelSize) { // GPU main kernel
    
    int startRawImageX = blockDim.x * blockIdx.x + threadIdx.x;
	int rawImageX = startRawImageX;
    int rawImageY = blockDim.y * blockIdx.y + threadIdx.y;

	for (; rawImageY < rawImageHeight; rawImageY += blockDim.y * gridDim.y)	{ // Work while this thread is inside the image data, otherwise bail out

		for (rawImageX = startRawImageX; rawImageX < rawImageWidth; rawImageX += blockDim.x * gridDim.x)	{ // proceed to X-grid exhaustion then consume Y

			float channelSum = 0;
			for (int ky = 0; ky < KERNEL_HEIGHT; ++ky) { // Kernel loop
				for (int kx = 0; kx < KERNEL_WIDTH; ++kx) {
    
					float pixelChannelValue;
					int requestedRawXpos = rawImageX - (KERNEL_WIDTH  / 2 + kx) * pixelSize; // Raw image relative coords
					int requestedRawYpos = rawImageY - (KERNEL_HEIGHT / 2 + ky); // Raw image relative coords

					// If the channel data requested is outside the image area simply 0-pad it
					if (requestedRawXpos < 0 || requestedRawYpos < 0 ||
						requestedRawXpos >= rawImageWidth || requestedRawYpos >= rawImageHeight)
						pixelChannelValue = 0;
					else
						// Load from global memory the raw image data (whatever channel this one is)
						pixelChannelValue = image[requestedRawYpos * rawImageWidth + requestedRawXpos];

					channelSum += constantKernel[ky * KERNEL_WIDTH + kx] * pixelChannelValue;
				}
			}

			// Store result back to global memory in the result image
			resultImage[rawImageY * rawImageWidth + rawImageX] = static_cast<unsigned char>(channelSum / constantKernelSum);
		}
	}
}


bool simpleGPUConvolution(PPMFile& imageFile) {

	hipEvent_t start, stop;   // Get up some metrics to measure GPU execution time
	hipEventCreate(&start);
	hipEventCreate(&stop);

    hipError_t err = hipSuccess;
    int pixelSize = sizeof(RGB);
    size_t rawImageSize = imageFile.width() * imageFile.height() * pixelSize;

    // Precompute kernel sum, this is faster on the CPU and not worth another kernel launch
    float kernelSum = std::accumulate(gaussianKernel.begin(), // Get the sum of the kernel elements
        gaussianKernel.end(), 0.0f,
        [](float sum, const float& elem) { 
                return sum + elem; 
		}
	);

	hipEventRecord(start, 0); // Start recording (including host<->device transfers)
    
    err = hipMemcpyToSymbol(HIP_SYMBOL(constantKernelSum), &kernelSum, sizeof(float));
    if (err != hipSuccess) {
        std::cout << "Failed to transfer memory to constant kernel variable: " << hipGetErrorString(err);
        return false;
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(constantKernel), gaussianKernel.data(), gaussianKernel.size() * sizeof(float));
    if (err != hipSuccess) {
        std::cout << "Failed to transfer memory to constant kernel variable: " << hipGetErrorString(err);
        return false;
    }

    unsigned char *d_image = nullptr, *d_resultImage = nullptr;
    err = hipMalloc((void **)&d_image, rawImageSize);
    if (err != hipSuccess) {
        std::cout << "Failed to allocate global memory: " << hipGetErrorString(err);
        return false;
    }
	err = hipMalloc((void **)&d_resultImage, rawImageSize);
    if (err != hipSuccess) {
        std::cout << "Failed to allocate global memory: " << hipGetErrorString(err);
        return false;
    }

    err = hipMemcpy(d_image, imageFile.getImageRawData(), rawImageSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cout << "Failed to transfer image to global memory: " << hipGetErrorString(err);
        return false;
    }

    dim3 gridSize(64, 64), blockSize(64, 16);
    convolutionGPUKernel<<<gridSize, blockSize>>> (d_image, d_resultImage, imageFile.width() * pixelSize, imageFile.height(), pixelSize);
    err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "Kernel launch failed" << hipGetErrorString(err);
        return false;
    }

    err = hipMemcpy(imageFile.getImageRawData(), d_resultImage, rawImageSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cout << "Transferring back device memory to host failed: " << hipGetErrorString(err);
        return false;
    }

	// Stop event
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop);

	std::cout << "GPU execution time: " << elapsedTime << " ms" << std::endl;
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

    err = hipFree(d_image);
    if (err != hipSuccess) {
        std::cout << "Freeing device memory failed: " << hipGetErrorString(err);
        return false;
    }
	err = hipFree(d_resultImage);
    if (err != hipSuccess) {
        std::cout << "Freeing device memory failed: " << hipGetErrorString(err);
        return false;
    }

    err = hipDeviceReset(); // This is not mandatory
    if (err != hipSuccess) {
        std::cout << "Failed to deinitialize the device: " << hipGetErrorString(err);
        return false;
    }

	return true;
}

int main(int argc, char* argv[]) {
    PPMFile originalPPM;
	originalPPM.readPPM("univpm.ppm");

    if (simpleGPUConvolution(originalPPM) == false)
		return -1;

	originalPPM.writePPM("univpmGPU.ppm");

	return 0;
}

